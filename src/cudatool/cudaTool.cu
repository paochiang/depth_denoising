#include "hip/hip_runtime.h"
#include "cudaTool.h"
#include <iostream>
#include <cstdio>
#include <fstream>
const int BLOCK_X = 16;
const int BLOCK_y = 16;
#define SafeCall(call)                                                         \
{                                                                              \
    const hipError_t error = call;                                            \
    if (error != hipSuccess)                                                  \
	    {                                                                      \
        fprintf(stderr, "Error: %s:%d, ", __FILE__, __LINE__);                 \
        fprintf(stderr, "code: %d, reason: %s\n", error,                       \
                hipGetErrorString(error));                                    \
        exit(1);                                                               \
	    }                                                                      \
}
bool cudaTool::InitCUDA(void)
{
	int count;	
	hipGetDeviceCount(&count);
	if (count == 0) 
	{
		printf("There is no device.\n");
		return false;
	}
	int i;
	for (i = 0; i<count; i++)
	{
		hipDeviceProp_t prop; 
		if (hipGetDeviceProperties(&prop, i) == hipSuccess)
		{
			if (prop.major >= 1)
			{
				break;
			}
		}
	}
	printf("Find %d GPUs\n", count);
	if (i == count)
	{
		printf("There is no device supporting CUDA 1.x\n");
		return false;
	}
	hipSetDevice(i);
	return true;
}
__global__ void sortDepth(float* D_cur_depth, float* D_sortedDepthData, int width, int height, int totalCount) {
	int dx = blockDim.x * blockIdx.x + threadIdx.x;
	int dy = blockDim.y * blockIdx.y + threadIdx.y;
	if (dx < width && dy < height) {
		unsigned int index = dy * width + dx;
		int i;
		for (i = totalCount - 2; i >= 0; i--) {
			if (D_sortedDepthData[index + width * height * i] > D_cur_depth[index]) {
				D_sortedDepthData[index + width * height * (i + 1)] = D_sortedDepthData[index + width * height * i];
			}
			else {
				break;
			}
		}
		D_sortedDepthData[index + width * height * (i + 1)] = D_cur_depth[index];
	}
}

__global__ void dealSortedDepth(float* D_sortedDepthData, char* D_mask, int width, int height, int totalCount, float threshold, float percent) {
	int dx = blockDim.x * blockIdx.x + threadIdx.x;
	int dy = blockDim.y * blockIdx.y + threadIdx.y;
	if (dx < width && dy < height) {
		unsigned int index = dy * width + dx;
		int offset = totalCount * percent;
		bool flag = false;
		float ave = 0;
		if (D_sortedDepthData[index + (totalCount - offset - 1) * width * height] - D_sortedDepthData[index + offset * width * height] > threshold * 3)
			flag = true;
		for (int i = offset; i < totalCount - offset - 1; i++) {
			if (flag)
				break;
			float cur = D_sortedDepthData[index + i * width * height];
			float behind = D_sortedDepthData[index + (i + 1) * width * height];
			if (behind - cur > threshold) {
				flag = true;
				break;
			}
			ave += cur;
		}
		if (flag)
			D_mask[index] = 0;
		else {
			ave += D_sortedDepthData[index + (totalCount - offset - 1) * width * height];
			ave /= (totalCount - 2 * offset);
			D_mask[index] = 1;
			D_sortedDepthData[index] = ave;
		}
	}

}

__global__ void dealUnSortedAllDepth(float* D_unSortedDepthData, char* D_mask, int width, int height, int totalCount, float threshold, float percent) {
	int dx = blockDim.x * blockIdx.x + threadIdx.x;
	int dy = blockDim.y * blockIdx.y + threadIdx.y;
	if (dx < width && dy < height) {
		unsigned int index = dy * width + dx;
		//sort
		for (int i = 0; i < totalCount - 1; i++) {
			float curMin = D_unSortedDepthData[index + i * width * height];
			int mark = i;
			for (int j = i + 1; j < totalCount; j++) {
				if (D_unSortedDepthData[index + j * width * height] < curMin) {
					curMin = D_unSortedDepthData[index + j * width * height];
					mark = j;
				}					
			}
			if (mark != i) {
				float t = D_unSortedDepthData[index + i * width * height];
				D_unSortedDepthData[index + i * width * height] = D_unSortedDepthData[index + mark * width * height];
				D_unSortedDepthData[index + mark * width * height] = t;
			}
		}

		//denoising
		int offset = totalCount * percent;
		bool flag = false;
		float ave = 0;
		if (D_unSortedDepthData[index + (totalCount - offset - 1) * width * height] - D_unSortedDepthData[index + offset * width * height] > threshold * 3)
			flag = true;
		for (int i = offset; i < totalCount - offset - 1; i++) {
			if (flag)
				break;
			float cur = D_unSortedDepthData[index + i * width * height];
			float behind = D_unSortedDepthData[index + (i + 1) * width * height];
			if (behind - cur > threshold) {
				flag = true;
				break;
			}
			ave += cur;
		}
		if (flag)
			D_mask[index] = 0;
		else {
			ave += D_unSortedDepthData[index + (totalCount - offset - 1) * width * height];
			ave /= (totalCount - 2 * offset);
			D_mask[index] = 1;
			D_unSortedDepthData[index] = ave;
		}
	}

}

std::vector<std::vector<float>> cudaTool::deelDethDenoise(float* curDepthData, float*& sortedDepthData, int width, int height, int totalCount, float threshold, float percent, bool isLast) {
	std::vector<std::vector<float>> res(width * height);	
	if (totalCount <= 1) {
		std::cout << "cudaTool:deelDethDenoise:totalCount should start at 2!" << std::endl;
		return res;
	}
	dim3 block(BLOCK_X, BLOCK_y);
	dim3 grid((width + block.x - 1) / block.x, (height + block.y - 1) / block.y);
	float* D_cur_depth;
	float* D_sortedDepthData;
	SafeCall(hipMalloc((void **)&D_cur_depth, sizeof(float) * width * height));	
	SafeCall(hipMemcpy(D_cur_depth, curDepthData, sizeof(float) * width * height, hipMemcpyHostToDevice));
	SafeCall(hipMalloc((void **)&D_sortedDepthData, sizeof(float) * width * height * totalCount));
	SafeCall(hipMemcpy(D_sortedDepthData, sortedDepthData, sizeof(float) * width * height *  (totalCount - 1), hipMemcpyHostToDevice));
	
	sortDepth <<<grid, block>>>(D_cur_depth, D_sortedDepthData, width, height, totalCount);

	if (isLast) 
	{
		char* D_mask;
		SafeCall(hipMalloc((void **)&D_mask, sizeof(char) * width * height));
		dealSortedDepth << <grid, block >> >(D_sortedDepthData, D_mask, width, height, totalCount, threshold, percent);
		char* mask = new char[width * height];
		float* denoiseRes = new float[width * height * totalCount];
		SafeCall(hipMemcpy(denoiseRes, D_sortedDepthData, sizeof(float) * width * height * totalCount, hipMemcpyDeviceToHost));
		SafeCall(hipMemcpy(mask, D_mask, sizeof(char) * width * height, hipMemcpyDeviceToHost));

		for (int r = 0; r < height; r++) 
		{
			for (int c = 0; c < width; c++) 
			{
				unsigned index = r * width + c;
				if (mask[index] == 1) 
				{
					res[index].push_back(denoiseRes[index]);
				}
				else {
					for (int i = 0; i < totalCount; i++) 
					{
						res[index].push_back(denoiseRes[index + i * width * height]);
					}
				}
			}
		}
		if (mask) {
			delete[] mask;
			mask = NULL;
		}
		if (denoiseRes) {
			delete[] denoiseRes;
			denoiseRes = NULL;
		}
		hipFree(D_mask);
	}
	else {
		if (sortedDepthData) {
			delete[] sortedDepthData;
			sortedDepthData = NULL;
		}
		sortedDepthData = new float[sizeof(float) * width * height * totalCount];
		SafeCall(hipMemcpy(sortedDepthData, D_sortedDepthData, sizeof(float) * width * height * totalCount, hipMemcpyDeviceToHost));
	}
	hipFree(D_cur_depth);
	hipFree(D_sortedDepthData);
	return res;
}

std::vector<std::vector<float>> cudaTool::deelAllDethDenoise(const std::vector<float>& unSortedDepthData, int width, int height, int totalCount, float threshold, float percent) {
	std::vector<std::vector<float>> res(width * height);
	if (totalCount <= 1) {
		std::cout << "cudaTool:deelDethDenoise:totalCount should start at 2!" << std::endl;
		return res;
	}
	dim3 block(BLOCK_X, BLOCK_y);
	dim3 grid((width + block.x - 1) / block.x, (height + block.y - 1) / block.y);
	float* allDepthData = new float[unSortedDepthData.size()];
	for (int i = 0; i < unSortedDepthData.size(); i++)
		allDepthData[i] = unSortedDepthData[i];
	float* D_unSortedDepthData;
	SafeCall(hipMalloc((void **)&D_unSortedDepthData, sizeof(float) * width * height * totalCount));
	SafeCall(hipMemcpy(D_unSortedDepthData, allDepthData, sizeof(float) * width * height *  totalCount, hipMemcpyHostToDevice));
	char* D_mask;
	SafeCall(hipMalloc((void **)&D_mask, sizeof(char) * width * height));

	dealUnSortedAllDepth << <grid, block >> >(D_unSortedDepthData, D_mask, width, height, totalCount, threshold, percent);

	char* mask = new char[width * height];
	SafeCall(hipMemcpy(allDepthData, D_unSortedDepthData, sizeof(float) * width * height * totalCount, hipMemcpyDeviceToHost));
	SafeCall(hipMemcpy(mask, D_mask, sizeof(char) * width * height, hipMemcpyDeviceToHost));

	for (int r = 0; r < height; r++)
	{
		for (int c = 0; c < width; c++)
		{
			unsigned index = r * width + c;
			if (mask[index] == 1)
			{
				res[index].push_back(allDepthData[index]);
			}
			else {
				for (int i = 0; i < totalCount; i++)
				{
					res[index].push_back(allDepthData[index + i * width * height]);
				}
			}
		}
	}
	if (mask) {
		delete[] mask;
		mask = NULL;
	}
	if (allDepthData) {
		delete[] allDepthData;
		allDepthData = NULL;
	}
	hipFree(D_mask);
	hipFree(D_unSortedDepthData);
	return res;
}
